#include "hip/hip_runtime.h"
#include "Bloom.h"

/**
* Allocates hiprand states.
*/
extern hiprandState* allocateCurandStates(int length){
	hiprandState* dev_states;
	hipError_t result = hipMalloc((void**)&dev_states,length*sizeof(int));
	if(result!=hipSuccess){
		printf("Could not allocate memory for the integers");
		return 0;
	}	 
	return dev_states;
}

/**
* Frees curandStates.
*/
extern hipError_t freeCurandStates(hiprandState* dev_states){
	return hipFree(dev_states);
}

/**
* Allocates an Integer array to the cuda device.
* @param array The array of integers being allocated.
* @param length The number of items in the array.
*/
int* allocateAndCopyIntegers(int* array,int length){
	int* dev_int;
	hipError_t result = hipMalloc((void**)&dev_int,length*sizeof(int));
	if(result!=hipSuccess){
		printf("Could not allocate memory for the integers");
		return 0;
	}	 
	result = hipMemcpy(dev_int,array,sizeof(int)*length,
		hipMemcpyHostToDevice);
	if(result!=hipSuccess){
		printf("Could not copy the integers ot the device \n");
		return 0;
	}	
	return dev_int; 
}

/**
* Frees Integers copied into a cuda array.
* @param dev_array The integers copied into the cuda array. 
*/
hipError_t freeIntegers(int* dev_array){
	return hipFree(dev_array);
}

/**
* Alloctes a Float array to the cuda device.
*/
float* allocateAndCopyFloats(float* array,int length){
	float* dev_float;
	hipError_t result = hipMalloc((void**)&dev_float,length*sizeof(float));
	if(result!=hipSuccess){
		printf("Could not allocate memory for the integers");
		return 0;
	}	 
	result = hipMemcpy(dev_float,array,sizeof(float)*length,
		hipMemcpyHostToDevice);
	if(result!=hipSuccess){
		printf("Could not copy the integers ot the device \n");
		return 0;
	}	
	return dev_float; 


}

/**
* Frees Floats copied into a cuda array.
*/
hipError_t freeFloats(float* dev_float){
	return hipFree(dev_float);
}


/**
* Allocates a character array to the cuda device.
* @param array
*/
char* allocateAndCopyChar(char* array,int length){
	char* dev_array;
	hipError_t result = hipMalloc((void**)&dev_array,length*sizeof(char));
	if(result!=hipSuccess){
		printf("Could not allocate the char array \n");
		return 0;
	}	 
	result = hipMemcpy(dev_array,array,sizeof(char)*length,
		hipMemcpyHostToDevice);
	if(result!=hipSuccess){	
		printf("Could copy the char array to the device \n");
		return 0;
	}	
	return dev_array; 
}

/**
* Fres Characters copied into a cuda array.
* @param dev_array The array being freed.
*/
hipError_t freeChars(char* dev_array){
	return hipFree(dev_array);
}

/**
* Copies a character array to the host.
* @param char* array The host array
* @param char8 dev_array The device array
* @param length The number of items being copied.
*/
hipError_t copyCharsToHost(char* array,char* dev_array,int length){
	return hipMemcpy(array,dev_array,sizeof(char)*length,hipMemcpyDeviceToHost);
}

/**
* Responsible for calculating the dimenions of the gpu layout being used.
* @param numWords
* @param numHash
* @param device
*/
dim3 calculateThreadDimensions(int numWords,int numHash,int device){
	if(numWords == 0 || numHash == 0){
		printf("Nothing to do \n");
		return dim3(0,0,0);
	}
	//Get the properties of the device the user selected.
	hipDeviceProp_t deviceProps;
  hipGetDeviceProperties(&deviceProps, device);
		
	//Firstly, solve for the max number of words that 
	//Can be processed in one thread block.
	int maxWordPerBlock = deviceProps.maxThreadsPerBlock/numHash;
	
	//Check to see if the user specified too many hash functions.
	if(maxWordPerBlock ==0){
		printf("Too many hash functions \n");
		return dim3(0,0); 
	}
	int wordsPerBlock = 32*(maxWordPerBlock/32);
	if(wordsPerBlock ==0)
		wordsPerBlock = maxWordPerBlock;
	dim3 threadDimensions(wordsPerBlock,numHash);
	return threadDimensions;
}

/**
* Responsible for calculating the thread dimensions of the gpu layout.
* @param threadDimensions the dimensions of the thread block.
* @param device The id of the device being used.
*/
dim3 calculateBlockDimensions(dim3 threadDimensions,int numWords,
	int device){
	if(numWords == 0){
		printf("Nothing to do \n");
		return dim3(0,0,0);
	}

	//Get the device information being used.
	hipDeviceProp_t deviceProps;
	hipGetDeviceProperties(&deviceProps,device);
	//Calculate the number of blocks needed to process all of the words.
	int numBlocksNeeded = numWords/threadDimensions.x;
	if(numWords%threadDimensions.x!=0)
		numBlocksNeeded++;
	//Hard coded due to hydra glitch.
	int maxGridSizeX = 65535;
	int numBlocksPerRow;	
	
	if(numBlocksNeeded<=maxGridSizeX)
		numBlocksPerRow = numBlocksNeeded;
	else{
		numBlocksPerRow = maxGridSizeX;
	}

	int numRows = numBlocksNeeded/numBlocksPerRow;
	if(numBlocksNeeded%numBlocksPerRow!=0){
		numRows++;
	}
	if(numRows>deviceProps.maxGridSize[1]){
		printf("Too many rows requested %i, \n",numRows);
		printf("Blocks Per Row %i \n",numBlocksPerRow);
		printf("threadDim: %i,%i \n",threadDimensions.x,threadDimensions.y); 
		return dim3(0,0);
	}
	
	return dim3(numBlocksPerRow,numRows);
}

/**
* Calculates the djb2 hash.
* @param str The string being hashed.
* @param start The starting point of the word in the array.
* @return Returns the djb2 hash in long format.
*/
__device__ unsigned long djb2Hash(unsigned char* str,int start){
	unsigned long hash = 5381;
	int c;
	while(str[start]!=','){
		c = (int)str[start];
		hash = ((hash<<5)+hash)+c;
		start++;
	}	
	return hash;
}

/**
* Calculates the sdbm hash.
* @param str The string being hashed.
* @param start The starting point of the word in the array.
* @return Returns the sdbm hash in long format.
*/
__device__ unsigned long sdbmHash(unsigned char* str,int start){
	unsigned long hash = 0;
	int c = 0;
	while(str[start]!=','){
		c = (int)str[start];
		hash = c+(hash<<6)+(hash<<16)-hash;
		start++;
	}
	return hash;
}

__device__ int calculateCurrentWord(){
	int numThreadsPrevRows = (blockDim.x*gridDim.x)*blockIdx.y+
														blockDim.x*blockIdx.x;
	return  threadIdx.x+numThreadsPrevRows;
}

__device__ int calculateIndex(char* dev_bloom,int size,char* dev_words,
	int wordStartingPosition){	

	unsigned long firstValue = djb2Hash((unsigned char*)dev_words,wordStartingPosition)%size;	
	unsigned long secondValue = sdbmHash((unsigned char*)dev_words,wordStartingPosition)%size;
	secondValue = (secondValue*threadIdx.y*threadIdx.y)%size;
	return (firstValue+secondValue)%size;

}

//Initialize the random values here...
__global__ void initRand(hiprandState* state,unsigned long seed,int numWords){
	int index = calculateCurrentWord();
	if(index>=numWords)
		return;
	hiprand_init(seed,index+threadIdx.y,0,&state[index+threadIdx.y]);
}

//Insert words into the gpu bloom.
__global__ void insertWordsGpuPBF(char* dev_bloom,
	int size,char* dev_words,int* dev_positions, int numWords,
	hiprandState* globalState,float prob){

	//Firstly, calculate the current index and the random probability.
	int word = calculateCurrentWord();
	if(word>=numWords)
		return;

	hiprandState localState = globalState[word+threadIdx.y];
	float random = hiprand_uniform(&localState);
	globalState[word+threadIdx.y] = localState;
	
	int index = calculateIndex(dev_bloom,size,dev_words,dev_positions[word]);
	//if it has NOT been set.
	if(dev_bloom[index]!=1){
		//If the probability is low enough...
		if(random<prob){
			dev_bloom[index] = 1;	
		}
	}		
}



/**
* Responsible for inserting words using the gpu.
* @param dev_bloom The bloom filter being used.
* @param dev_size The size of the bloom filter being used.
* @param dev_words The words being inserted.
* @param dev_positions The starting positions of the words.
* @param dev_numWords The number of words being inserted.
*/
__global__ void insertWordsGpu(char* dev_bloom,int size,char* dev_words,
	int* dev_positions,int numWords){
	int currentWord = calculateCurrentWord();
	if(currentWord>=numWords)
		return;
	int wordStartingPosition = dev_positions[currentWord]; 	
	int setIdx = calculateIndex(dev_bloom,size,dev_words,
		wordStartingPosition);
	dev_bloom[setIdx]=1;
}

/**
* Responsible for querying words using the gpu.
*/
__global__ void queryWordsGpu(char* dev_bloom,int size,char* dev_words,
	int* dev_positions,char* dev_results,int numWords){

	int currentWord = calculateCurrentWord();
	if(currentWord>=numWords)
		return;
	int wordStartingPosition = dev_positions[currentWord]; 
	int getIdx = calculateIndex(dev_bloom,size,dev_words,
		wordStartingPosition);
	__syncthreads();
	
	if(dev_bloom[getIdx]==0){
		dev_results[currentWord]=0;
	}
}

/**
* Responsible for inserting words into the bloom filter.
*/
hipError_t insertWords(char* dev_bloom,int size,char* words,
	int* offsets,int numWords,int numBytes,int numHashes,int device){

	dim3 threadDimensions = calculateThreadDimensions(numWords,numHashes,device);
	dim3 blockDimensions = calculateBlockDimensions(threadDimensions,numWords,
		device);	


	int* dev_offsets = allocateAndCopyIntegers(offsets,numWords);
	if(!dev_offsets){
		return hipGetLastError();
	}
		
	char* dev_words = allocateAndCopyChar(words,numBytes);
	if(!dev_words){
		return hipGetLastError();
	}

	//Actually insert the words.
	insertWordsGpu<<<blockDimensions,threadDimensions>>>(dev_bloom,size
		,dev_words,dev_offsets,numWords);
	hipDeviceSynchronize();
	freeChars(dev_words);
	freeIntegers(dev_offsets);

	//Check for errorrs...
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess){
		printf("%s \n",hipGetErrorString(error));
		printf("Dimensions calculated: \n");
		printf("threadDim: %i,%i \n",threadDimensions.x,threadDimensions.y); 
		printf("BlockDim: %i,%i \n",blockDimensions.x,blockDimensions.y);
		return error;
	}
	return hipSuccess;			 				
}

/**
* Responsible for uerying words inserted into the bloom filter
*/
hipError_t queryWords(char* dev_bloom,int size,char* words,
	int* offsets,int numWords,int numBytes,int numHashes,int device,
	char* results){

	dim3 threadDimensions = calculateThreadDimensions(numWords,numHashes,device);
	dim3 blockDimensions = calculateBlockDimensions(threadDimensions,numWords,
		device);
		

	int* dev_offsets = allocateAndCopyIntegers(offsets,numWords);
	if(!dev_offsets){
		return hipGetLastError();
	}
		
	char* dev_words = allocateAndCopyChar(words,numBytes);
	if(!dev_words){
		return hipGetLastError();
	}

	char* dev_results = allocateAndCopyChar(results,numWords);
	if(!dev_results){
		return hipGetLastError();
	}

	//Actually query the words.
	queryWordsGpu<<<blockDimensions,threadDimensions>>>(dev_bloom,size
		,dev_words,dev_offsets,dev_results,numWords);
	hipDeviceSynchronize();
	copyCharsToHost(results,dev_results,numWords);
	freeChars(dev_words);
	freeChars(dev_results);
	freeIntegers(dev_offsets);
	
	//Check for errorrs...
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess){
		printf("%s \n",hipGetErrorString(error));
		printf("Dimensions calculated: \n");
		printf("threadDim: %i,%i \n",threadDimensions.x,threadDimensions.y); 
		printf("BlockDim: %i,%i \n",blockDimensions.x,blockDimensions.y);
		return error;
	}
	
	return hipSuccess;			 				
}

/**
* Responsible for inserting words into the PBF.
*/
hipError_t insertWordsPBF(char* dev_bloom,int size,char* words,
	int* offsets,int numWords,int numBytes,int numHashes,int device,float prob){

	//Calculate the dimensions and allocate the gpu memory required.
	dim3 threadDimensions = calculateThreadDimensions(numWords,numHashes,device);
	dim3 blockDimensions = calculateBlockDimensions(threadDimensions,numWords,
		device);

	int* dev_offsets = allocateAndCopyIntegers(offsets,numWords);
	if(!dev_offsets){
		printf("Could not allocate the offsets \n");
		return hipGetLastError();
	}
		
	char* dev_words = allocateAndCopyChar(words,numBytes);
	if(!dev_words){
		printf("Could not allocate the words \n");;
		return hipGetLastError();
	}

	//Allocate the hiprand states.
	//A new random value for each word.
	hiprandState* dev_states = allocateCurandStates(numWords*numHashes);	
	if(!dev_states){
		return hipGetLastError();
	}

	//Seed the random values...	
	initRand<<<blockDimensions,threadDimensions>>>(dev_states,
		time(0),numWords);
	
	//Make sure we could initialize rand.
	hipDeviceSynchronize();
	hipError_t error = hipGetLastError();
	if(error!=hipSuccess){
		printf("could not initialize rand \n");
		printf("%s \n",hipGetErrorString(error));
		printf("Blocks Per Row %i,%i \n",blockDimensions.x,blockDimensions.y);
		printf("threadDim: %i,%i \n",threadDimensions.x,threadDimensions.y); 
		return error;
	}	

	insertWordsGpuPBF<<<blockDimensions,threadDimensions>>>(dev_bloom,size,
		dev_words,dev_offsets,numWords,dev_states,prob);	
	//Make sure we could generate random numbers.
	hipDeviceSynchronize();
	error = hipGetLastError();
	if(error!=hipSuccess){
		printf("could not generate \n");
		printf("%s \n",hipGetErrorString(error));
		printf("Blocks Per Row %i,%i \n",blockDimensions.x,blockDimensions.y);
		printf("threadDim: %i,%i \n",threadDimensions.x,threadDimensions.y); 
		return error;
	}	

	freeChars(dev_words);
	freeIntegers(dev_offsets);
				
	//Free the cuda random states.
	freeCurandStates(dev_states);
	return hipSuccess;
}

